#include "hip/hip_runtime.h"
#include <stdio.h>
#include "support.h"
#include "kernel.cu"
#define BLOCK_SIZE (TILE_SIZE+FILTER_SIZE-1)

void printMatrix(Matrix matrix) {
    
    for (int i = 0; i < matrix.height; i++) {
        for (int j = 0; j < matrix.width; j++) {
            printf("%.2f ", matrix.elements[i*matrix.width + j]);
        }
        printf("\n");
    }

}

int main(int argc, char* argv[])
{
    Timer timer;

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    Matrix M_h, N_h, P_h; // M: filter, N: input image, P: output image
    Matrix N_d, P_d;
    unsigned imageHeight, imageWidth;
    hipError_t cuda_ret;
    dim3 dim_grid, dim_block;

    /* Read image dimensions */
    if (argc == 3) {
        imageHeight = atoi(argv[1]);
        imageWidth = atoi(argv[2]);
    } else {
        printf("\n    Invalid input parameters!"
	   "\n    Usage: ./convolution <m> <n>  # Image is m x n"
           "\n");
        exit(0);
    }

    /* Allocate host memory */
    M_h = allocateMatrix(FILTER_SIZE, FILTER_SIZE);
    N_h = allocateMatrix(imageHeight, imageWidth);
    P_h = allocateMatrix(imageHeight, imageWidth);

    /* Initialize filter and images */
    initMatrix(M_h);
    initMatrix(N_h);

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    Image: %u x %u\n", imageHeight, imageWidth);
    printf("    Mask: %u x %u\n", FILTER_SIZE, FILTER_SIZE);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    N_d = allocateDeviceMatrix(imageHeight, imageWidth);
    P_d = allocateDeviceMatrix(imageHeight, imageWidth);

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    /* Copy image to device global memory */
    //INSERT CODE HERE
    hipMemcpy(N_d.elements, N_h.elements, imageHeight*imageHeight*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(P_d.elements, P_h.elements, imageHeight*imageHeight*sizeof(float), hipMemcpyHostToDevice);

    /* Copy mask to device constant memory */
    //INSERT CODE HERE
    hipMemcpyToSymbol(HIP_SYMBOL(Mc), M_h.elements, FILTER_SIZE*FILTER_SIZE*sizeof(float));
    
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));


    /*Launch kernel ---------------------------------------------------------*/
    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);
    //INSERT CODE HERE
    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
    dim3 dimGrid(ceil(N_h.width/float(TILE_SIZE)), ceil(N_h.height/float(TILE_SIZE)),1); 
    convolution<<<dimGrid,dimBlock>>>(N_d,P_d);

    
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch/execute kernel");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy data from device to host ----------------------------------------
    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);

    copyFromDeviceMatrix(P_h, P_d);

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Print -----------------------------------------------------------------
    // Print N_d
    printf("\nContents of N_h (input image):\n");
    printMatrix(N_h);
    // Print Mc
    printf("\nContents of Mc (filter):\n");
    printMatrix(M_h);
    // Print P_d
    printf("\nContents of P_h (output image):\n");
    printMatrix(P_h);

    // Verify correctness -----------------------------------------------------

    printf("Verifying results..."); fflush(stdout);

    verify(M_h, N_h, P_h);

    // Free memory ------------------------------------------------------------

     freeMatrix(M_h);
     freeMatrix(N_h);
     freeMatrix(P_h);
     freeDeviceMatrix(N_d);
     freeDeviceMatrix(P_d);

     return 0;
}

